#include <CUDA/PointCloudBuffers.cuh>

//void PointCloudBuffers::Initialize(unsigned int numberOfPoints, bool isHostBuffer)
//{
//	this->isHostBuffer = isHostBuffer;
//	this->numberOfPoints = numberOfPoints;
//
//	if (isHostBuffer)
//	{
//		positions = new Eigen::Vector3f[numberOfPoints];
//		normals = new Eigen::Vector3f[numberOfPoints];
//		colors = new Eigen::Vector4b[numberOfPoints];
//	}
//	else
//	{
//		hipMalloc(&positions, sizeof(Eigen::Vector3f) * numberOfPoints);
//		hipMalloc(&normals, sizeof(Eigen::Vector3f) * numberOfPoints);
//		hipMalloc(&colors, sizeof(Eigen::Vector4b) * numberOfPoints);
//	}
//}
//
//void PointCloudBuffers::Terminate()
//{
//	if (isHostBuffer)
//	{
//		if (0 < numberOfPoints)
//		{
//			delete[] positions;
//			delete[] normals;
//			delete[] colors;
//		}
//	}
//	else
//	{
//		if (0 < numberOfPoints)
//		{
//			hipFree(positions);
//			hipFree(normals);
//			hipFree(colors);
//		}
//	}
//}
//
//void PointCloudBuffers::CopyTo(PointCloudBuffers& other)
//{
//	if (isHostBuffer && other.isHostBuffer)
//	{
//		other.numberOfPoints = numberOfPoints;
//		other.aabb = aabb;
//		memcpy(other.positions, positions, sizeof(Eigen::Vector3f) * numberOfPoints);
//		memcpy(other.normals, normals, sizeof(Eigen::Vector3f) * numberOfPoints);
//		memcpy(other.colors, colors, sizeof(Eigen::Vector4b) * numberOfPoints);
//	}
//	else if (false == isHostBuffer && other.isHostBuffer)
//	{
//		other.numberOfPoints = numberOfPoints;
//		other.aabb = aabb;
//		hipMemcpy(other.positions, positions, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyDeviceToHost);
//		hipMemcpy(other.normals, normals, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyDeviceToHost);
//		hipMemcpy(other.colors, colors, sizeof(Eigen::Vector4b) * numberOfPoints, hipMemcpyDeviceToHost);
//	}
//	else if (isHostBuffer && false == other.isHostBuffer)
//	{
//		other.numberOfPoints = numberOfPoints;
//		other.aabb = aabb;
//		hipMemcpy(other.positions, positions, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyHostToDevice);
//		hipMemcpy(other.normals, normals, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyHostToDevice);
//		hipMemcpy(other.colors, colors, sizeof(Eigen::Vector4b) * numberOfPoints, hipMemcpyHostToDevice);
//	}
//	else if (false == isHostBuffer && false == other.isHostBuffer)
//	{
//		other.numberOfPoints = numberOfPoints;
//		other.aabb = aabb;
//		hipMemcpy(other.positions, positions, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyDeviceToDevice);
//		hipMemcpy(other.normals, normals, sizeof(Eigen::Vector3f) * numberOfPoints, hipMemcpyDeviceToDevice);
//		hipMemcpy(other.colors, colors, sizeof(Eigen::Vector4b) * numberOfPoints, hipMemcpyDeviceToDevice);
//	}
//}
