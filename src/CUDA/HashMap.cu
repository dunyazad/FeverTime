#include "hip/hip_runtime.h"
#include <CUDA/HashMap.cuh>

void HashMap::Initialize()
{
	hipMalloc(&info.d_hashTable, sizeof(HashMapVoxel) * info.capacity);
	hipMemset(info.d_hashTable, 0, sizeof(HashMapVoxel) * info.capacity);

	hipMalloc(&info.d_numberOfOccupiedVoxels, sizeof(unsigned int));
	hipMemset(info.d_numberOfOccupiedVoxels, 0, sizeof(unsigned int));

	hipMalloc(&info.d_occupiedVoxelIndices, sizeof(int3) * info.capacity);
	hipMemset(info.d_occupiedVoxelIndices, 0, sizeof(int3) * info.capacity);

	info.labelCounter.Initialize(info.capacity);
	info.subLabelCounter.Initialize(info.capacity);
}

void HashMap::Terminate()
{
	hipFree(info.d_hashTable);
	hipFree(info.d_numberOfOccupiedVoxels);
	hipFree(info.d_occupiedVoxelIndices);

	info.labelCounter.Terminate();
	info.subLabelCounter.Terminate();
}

__global__ void Kernel_InsertPoints(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto p = buffers.positions[idx];
	auto n = buffers.normals[idx].normalized();
	auto c = buffers.colors[idx];

	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));

	size_t h = voxel_hash(coord, info.capacity);
	for (int i = 0; i < info.maxProbe; ++i) {
		size_t slot = (h + i) % info.capacity;
		int prev = atomicCAS(&(info.d_hashTable[slot].label), 0, slot);

		if (prev == 0) {
			// ���ο� ���Կ� ����
			info.d_hashTable[slot].coord = coord;
			info.d_hashTable[slot].position = Eigen::Vector3f((float)coord.x * info.voxelSize, (float)coord.y * info.voxelSize, (float)coord.z * info.voxelSize);
			info.d_hashTable[slot].normal = n;
			info.d_hashTable[slot].color = c;
			info.d_hashTable[slot].pointCount = 1;

			auto oldIndex = atomicAdd(info.d_numberOfOccupiedVoxels, 1);
			info.d_occupiedVoxelIndices[oldIndex] = coord;
			return;
		}
		else {
			int3 existing = info.d_hashTable[slot].coord;
			if (existing.x == coord.x && existing.y == coord.y && existing.z == coord.z) {
				info.d_hashTable[slot].normal += n;
				info.d_hashTable[slot].color = c;
				info.d_hashTable[slot].pointCount++;
				return;
			}
		}
	}
}

void HashMap::InsertPoints(PointCloudBuffers buffers)
{
	unsigned int blockSize = 256;
	unsigned int gridOccupied = (buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_InsertPoints << <gridOccupied, blockSize >> > (info, buffers);

	hipDeviceSynchronize();

	hipMemcpy(&info.h_numberOfOccupiedVoxels, info.d_numberOfOccupiedVoxels, sizeof(unsigned int), hipMemcpyDeviceToHost);
}

__global__ void Kernel_CountLabels(HashMapInfo info)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[idx];
	auto voxelSlot = GetHashMapVoxelSlot(info, coord);
	if (UINT64_MAX == voxelSlot) return;

	auto voxel = GetHashMapVoxel(info, voxelSlot);
	if (nullptr == voxel) return;

	info.labelCounter.IncreaseCount(voxel->label);
	info.subLabelCounter.IncreaseCount(voxel->subLabel);
}

void HashMap::CountLabels()
{
	info.labelCounter.Clear();
	info.subLabelCounter.Clear();

	//info.labelCounter.Resize(info.capacity * 2);
	//info.subLabelCounter.Resize(info.capacity * 2);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (info.h_numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_CountLabels << <gridOccupied, blockSize >> > (info);

	hipDeviceSynchronize();
}

__global__ void Kernel_Serialize_HashMap(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[idx];
	size_t h = voxel_hash(coord, info.capacity);

	for (unsigned int i = 0; i < info.maxProbe; ++i)
	{
		size_t slot = (h + i) % info.capacity;
		auto& voxel = info.d_hashTable[slot];

		if (0 == voxel.label) return;

		if (voxel.coord.x == coord.x &&
			voxel.coord.y == coord.y &&
			voxel.coord.z == coord.z)
		{
			buffers.positions[idx] = voxel.position;
			buffers.normals[idx] = (voxel.normal / (float)voxel.pointCount).normalized();
			buffers.colors[idx] = voxel.color / voxel.pointCount;
			return;
		}
	}
}

void HashMap::SerializeToPLY(const std::string& filename)
{
	PLYFormat ply;

	unsigned int numberOfOccupiedVoxels = info.h_numberOfOccupiedVoxels;

	PointCloudBuffers d_buffers;
	d_buffers.Initialize(numberOfOccupiedVoxels, false);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_Serialize_HashMap << <gridOccupied, blockSize >> > (info, d_buffers);

	hipDeviceSynchronize();

	PointCloudBuffers h_buffers;
	h_buffers.Initialize(numberOfOccupiedVoxels, true);

	d_buffers.CopyTo(h_buffers);

	for (size_t i = 0; i < numberOfOccupiedVoxels; i++)
	{
		auto& p = h_buffers.positions[i];
		auto& n = h_buffers.normals[i];
		auto& c = h_buffers.colors[i];

		ply.AddPoint(p.x(), p.y(), p.z());
		ply.AddNormal(n.x(), n.y(), n.z());
		ply.AddColor(
			fminf(1.0f, fmaxf(0.0f, c.x())) / 255.0f,
			fminf(1.0f, fmaxf(0.0f, c.y())) / 255.0f,
			fminf(1.0f, fmaxf(0.0f, c.z())) / 255.0f
		);
	}

	ply.Serialize(filename);

	d_buffers.Terminate();
	h_buffers.Terminate();
}

__device__ size_t GetHashMapVoxelSlot(HashMapInfo& info, int3 coord)
{
	size_t h = voxel_hash(coord, info.capacity);
	for (int i = 0; i < info.maxProbe; ++i) {
		size_t slot = (h + i) % info.capacity;

		if (1 == info.d_hashTable[slot].deleted) continue;

		if (info.d_hashTable[slot].coord.x == coord.x &&
			info.d_hashTable[slot].coord.y == coord.y &&
			info.d_hashTable[slot].coord.z == coord.z)
		{
			return slot;
		}
	}

	return UINT64_MAX;
}

__device__ HashMapVoxel* GetHashMapVoxel(HashMapInfo& info, size_t slot)
{
	if (UINT64_MAX == slot) return nullptr;
	return &(info.d_hashTable[slot]);
}

__device__ size_t InsertHashMapVoxel(HashMapInfo& info, int3 coord)
{
	size_t h = voxel_hash(coord, info.capacity);

	for (int i = 0; i < info.maxProbe; ++i)
	{
		size_t slot = (h + i) % info.capacity;
		HashMapVoxel* voxel = &info.d_hashTable[slot];

		// �� �����̸� ���������� ���� �õ�
		int prev = atomicCAS(&(voxel->label), 0, slot);
		if (prev == 0)
		{
			// ���� ����, �ʱ�ȭ ����
			voxel->coord = coord;
			voxel->position = Eigen::Vector3f(coord.x * info.voxelSize, coord.y * info.voxelSize, coord.z * info.voxelSize);
			voxel->pointCount = 0;
			voxel->neighborCount = 0;
			voxel->emptyNeighborCount = 0;
			voxel->normal = Eigen::Vector3f::Zero();
			voxel->gradient = Eigen::Vector3f::Zero();
			voxel->divergence = 0.0f;
			voxel->colorDistance = 0.0f;
			voxel->normalDiscontinue = 0;
			voxel->color = Eigen::Vector4b(255, 255, 255, 255);
			voxel->deleted = 0;

			unsigned int index = atomicAdd(info.d_numberOfOccupiedVoxels, 1);
			info.d_occupiedVoxelIndices[index] = coord;

			return slot;
		}
		else
		{
			// �̹� �����ϴ� ��� ���� ��ǥ���� Ȯ��
			if (voxel->coord.x == coord.x &&
				voxel->coord.y == coord.y &&
				voxel->coord.z == coord.z)
			{
				return slot;
			}
		}
	}

	// ���� ����
	return UINT64_MAX;
}

__device__ bool DeleteHashMapVoxel(HashMapInfo& info, int3 coord)
{
	size_t h = voxel_hash(coord, info.capacity);
	for (int i = 0; i < info.maxProbe; ++i)
	{
		size_t slot = (h + i) % info.capacity;
		HashMapVoxel* voxel = &info.d_hashTable[slot];

		// ��ǥ ��ġ �� ���� ó��
		if (voxel->coord.x == coord.x &&
			voxel->coord.y == coord.y &&
			voxel->coord.z == coord.z &&
			voxel->deleted == 0)
		{
			voxel->deleted = 1;
			voxel->label = 0;  // �ʿ信 ���� reset
			voxel->subLabel = 0;
			voxel->reservedToDeleted = 0;
			return true;
		}
	}

	return false;
}
