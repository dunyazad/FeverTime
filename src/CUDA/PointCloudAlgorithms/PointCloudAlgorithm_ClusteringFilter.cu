#include "hip/hip_runtime.h"
#include <CUDA/PointCloudAlgorithms/PointCloudAlgorithm_ClusteringFilter.cuh>

#include <CUDA/PCD.cuh>

PointCloudAlgorithm_ClusteringFilter::PointCloudAlgorithm_ClusteringFilter()
{

}

PointCloudAlgorithm_ClusteringFilter::~PointCloudAlgorithm_ClusteringFilter()
{

}

//vector<uint3> PointCloud::Clustering(float normalDegreeThreshold)
//{
//	nvtxRangePushA("Clustering");
//
//	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;
//	
//	vector<uint3> labels;
//
//	{
//		unsigned int blockSize = 256;
//		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;
//
//		Kernel_ClearLabels << <gridOccupied, blockSize >> > (hashmap.info);
//
//		hipDeviceSynchronize();
//	}
//
//	{
//		unsigned int blockSize = 256;
//		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;
//
//		Kernel_InterVoxelHashMerge26Way << <gridOccupied, blockSize >> > (hashmap.info, normalDegreeThreshold);
//
//		hipDeviceSynchronize();
//	}
//
//	{
//		unsigned int blockSize = 256;
//		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;
//
//		Kernel_CompressVoxelHashLabels << <gridOccupied, blockSize >> > (hashmap.info);
//
//		hipDeviceSynchronize();
//	}
//
//	hashmap.CountLabels();
//
//	labels.resize(h_buffers.numberOfPoints);
//	{
//		uint3* d_labels = nullptr;
//		hipMalloc(&d_labels, sizeof(uint3) * h_buffers.numberOfPoints);
//
//		unsigned int blockSize = 256;
//		unsigned int gridOccupied = (h_buffers.numberOfPoints + blockSize - 1) / blockSize;
//
//		Kernel_GetLabels << <gridOccupied, blockSize >> > (hashmap.info, d_buffers.positions, d_labels, h_buffers.numberOfPoints);
//
//		hipDeviceSynchronize();
//
//		hipMemcpy(labels.data(), d_labels, sizeof(unsigned int) * h_buffers.numberOfPoints, hipMemcpyDeviceToHost);
//
//		hipFree(d_labels);
//	}
//
//	nvtxRangePop();
//
//	return labels;
//}

//void PointCloud::SerializeColoringByLabel(PointCloudBuffers& d_tempBuffers)
//{
//	d_buffers.CopyTo(d_tempBuffers);
//
//	unsigned int blockSize = 256;
//	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;
//
//	Kernel_SerializeColoringByLabel << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);
//
//	hipDeviceSynchronize();
//}
//
//__global__ void Kernel_SerializeColoringBySubLabel(HashMapInfo info, PointCloudBuffers buffers)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if (idx >= buffers.numberOfPoints) return;
//
//	auto& p = buffers.positions[idx];
//	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
//	size_t h = voxel_hash(coord, info.capacity);
//
//	for (unsigned int i = 0; i < info.maxProbe; ++i)
//	{
//		size_t slot = (h + i) % info.capacity;
//		auto& voxel = info.d_hashTable[slot];
//
//		if (0 == voxel.subLabel) return;
//
//		if (voxel.coord.x == coord.x &&
//			voxel.coord.y == coord.y &&
//			voxel.coord.z == coord.z)
//		{
//			buffers.positions[idx] = voxel.position;
//			buffers.normals[idx] = voxel.normal;
//
//			float r = hashToFloat(voxel.subLabel * 3 + 0);
//			float g = hashToFloat(voxel.subLabel * 3 + 1);
//			float b = hashToFloat(voxel.subLabel * 3 + 2);
//
//			auto subLabelCount = info.labelCounter.GetCount(voxel.subLabel);
//			if (100000 > subLabelCount)
//			{
//				buffers.colors[idx] = Eigen::Vector4b(r * 255.0f, g * 255.0f, b * 255.0f, 0);
//			}
//			else
//			{
//				buffers.colors[idx] = Eigen::Vector4b(r * 255.0f, g * 255.0f, b * 255.0f, 255);
//			}
//
//			return;
//		}
//	}
//}
//
//void PointCloud::SerializeColoringBySubLabel(PointCloudBuffers& d_tempBuffers)
//{
//	d_buffers.CopyTo(d_tempBuffers);
//
//	unsigned int blockSize = 256;
//	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;
//
//	Kernel_SerializeColoringBySubLabel << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);
//
//	hipDeviceSynchronize();
//}

__global__ void Kernel_SerializeFilteringColoringByLabel(
	HashMapInfo info,
	float3* positions,
	float3* normals,
	uchar4* colors,
	size_t numberOfPoints,
	bool applyColor)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numberOfPoints) return;

	auto& p = positions[idx];
	int3 coord = make_int3(floorf(p.x / info.voxelSize), floorf(p.y / info.voxelSize), floorf(p.z / info.voxelSize));

	auto slot = GetHashMapVoxelSlot(info, coord);
	if (INVALID_VOXEL_SLOT == slot) return;

	auto voxel = GetHashMapVoxel(info, slot);
	if (INVALID_VOXEL == voxel) return;

	if (0 == voxel->label) return;

	if (voxel->coord.x == coord.x &&
		voxel->coord.y == coord.y &&
		voxel->coord.z == coord.z)
	{
		if (applyColor)
		{
			normals[idx].x = voxel->normal.x() / (float)voxel->pointCount;
			normals[idx].y = voxel->normal.y() / (float)voxel->pointCount;
			normals[idx].z = voxel->normal.z() / (float)voxel->pointCount;

			float r = hashToFloat(voxel->label * 3 + 0);
			float g = hashToFloat(voxel->label * 3 + 1);
			float b = hashToFloat(voxel->label * 3 + 2);

			colors[idx] = make_uchar4(r * 255.0f, g * 255.0f, b * 255.0f, 255);
		}

		auto labelCount = info.labelCounter.GetCount(voxel->label);
		
		if (20000 > labelCount)
		{
			positions[idx].x = FLT_MAX;
			positions[idx].y = FLT_MAX;
			positions[idx].z = FLT_MAX;
		}

		//{
		//	buffers.colors[idx] = Eigen::Vector4b(r * 255.0f, g * 255.0f, b * 255.0f, 0);
		//}
		//else
		//{
		//	buffers.colors[idx] = Eigen::Vector4b(r * 255.0f, g * 255.0f, b * 255.0f, 255);
		//}
	}
}

void PointCloudAlgorithm_ClusteringFilter::RunAlgorithm(DevicePointCloud* pointCloud)
{
	nvtxRangePushA("Clustering");

	unsigned int numberOfOccupiedVoxels = pointCloud->GetHashMap().info.h_numberOfOccupiedVoxels;

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_ClearLabels << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_InterVoxelHashMerge26Way << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info, angleThreshold);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_CompressVoxelHashLabels << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info);

		hipDeviceSynchronize();
	}

	pointCloud->GetHashMap().CountLabels();

	{
		unsigned int numberOfPoints = pointCloud->GetNumberOfElements();

		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfPoints + blockSize - 1) / blockSize;

		auto positions = thrust::raw_pointer_cast(pointCloud->GetPositions().data());
		auto normals = thrust::raw_pointer_cast(pointCloud->GetNormals().data());
		auto colors = thrust::raw_pointer_cast(pointCloud->GetColors().data());

		Kernel_SerializeFilteringColoringByLabel << <gridOccupied, blockSize >> > (
			pointCloud->GetHashMap().info, positions, normals, colors, numberOfPoints, applyColor);

		hipDeviceSynchronize();
	}

	/*

	labels.resize(h_buffers.numberOfPoints);
	{
		uint3* d_labels = nullptr;
		hipMalloc(&d_labels, sizeof(uint3) * h_buffers.numberOfPoints);

		unsigned int blockSize = 256;
		unsigned int gridOccupied = (h_buffers.numberOfPoints + blockSize - 1) / blockSize;

		Kernel_GetLabels << <gridOccupied, blockSize >> > (hashmap.info, d_buffers.positions, d_labels, h_buffers.numberOfPoints);

		hipDeviceSynchronize();

		hipMemcpy(labels.data(), d_labels, sizeof(unsigned int) * h_buffers.numberOfPoints, hipMemcpyDeviceToHost);

		hipFree(d_labels);
	}*/

	nvtxRangePop();
}

void PointCloudAlgorithm_ClusteringFilter::RunAlgorithm(HostPointCloud* pointCloud)
{
	nvtxRangePushA("Clustering");

	unsigned int numberOfOccupiedVoxels = pointCloud->GetHashMap().info.h_numberOfOccupiedVoxels;

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_ClearLabels << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_InterVoxelHashMerge26Way << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info, angleThreshold);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_CompressVoxelHashLabels << <gridOccupied, blockSize >> > (pointCloud->GetHashMap().info);

		hipDeviceSynchronize();
	}

	pointCloud->GetHashMap().CountLabels();

	{
		unsigned int numberOfPoints = pointCloud->GetNumberOfElements();

		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfPoints + blockSize - 1) / blockSize;

		thrust::device_vector<float3> h_positions(pointCloud->GetPositions());
		thrust::device_vector<float3> h_normals(pointCloud->GetNormals());
		thrust::device_vector<uchar4> h_colors(pointCloud->GetColors());

		auto positions = thrust::raw_pointer_cast(h_positions.data());
		auto normals = thrust::raw_pointer_cast(h_normals.data());
		auto colors = thrust::raw_pointer_cast(h_colors.data());

		Kernel_SerializeColoringByLabel << <gridOccupied, blockSize >> > (
			pointCloud->GetHashMap().info, positions, normals, colors, numberOfPoints);

		hipDeviceSynchronize();
	}
}

void PointCloudAlgorithm_ClusteringFilter::IncreaseParameter()
{
	angleThreshold += 1.0f;
}

void PointCloudAlgorithm_ClusteringFilter::DecreaseParameter()
{
	angleThreshold -= 1.0f;
}
