#include "hip/hip_runtime.h"
#include <PointCloud.cuh>

#include <Serialization.hpp>

PointCloud::PointCloud()
{
}

PointCloud::~PointCloud()
{
}

void PointCloud::Initialize(unsigned int numberOfPoints)
{
	h_buffers.Initialize(numberOfPoints, true);
	d_buffers.Initialize(numberOfPoints, false);

	hashmap.Initialize();
}

void PointCloud::Terminate()
{
	h_buffers.Terminate();
	d_buffers.Terminate();

	hashmap.Terminate();
}

void PointCloud::HtoD()
{
	h_buffers.CopyTo(d_buffers);
}

void PointCloud::DtoH()
{
	d_buffers.CopyTo(h_buffers);
}

bool PointCloud::LoadFromPLY(const std::string& filename)
{
	PLYFormat ply;
	if (false == ply.Deserialize(filename))
	{
		return false;
	}

	Initialize(ply.GetPoints().size() / 3);

	for (size_t i = 0; i < ply.GetPoints().size() / 3; i++)
	{
		auto x = ply.GetPoints()[i * 3];
		auto y = ply.GetPoints()[i * 3 + 1];
		auto z = ply.GetPoints()[i * 3 + 2];

		auto nx = ply.GetNormals()[i * 3];
		auto ny = ply.GetNormals()[i * 3 + 1];
		auto nz = ply.GetNormals()[i * 3 + 2];

		auto r = ply.GetColors()[i * 3];
		auto g = ply.GetColors()[i * 3 + 1];
		auto b = ply.GetColors()[i * 3 + 2];

		h_buffers.positions[i] = Eigen::Vector3f(x, y, z);
		h_buffers.normals[i] = Eigen::Vector3f(nx, ny, nz);
		h_buffers.colors[i] = Eigen::Vector3b(r * 255.0f, g * 255.0f, b * 255.0f);

		h_buffers.aabb.extend(Eigen::Vector3f(x, y, z));
	}

	HtoD();

	hashmap.InsertPoints(d_buffers);

	return true;
}

bool PointCloud::LoadFromPLY(const string& filename, const Eigen::AlignedBox3f& roi)
{
	PLYFormat ply;
	if (false == ply.Deserialize(filename))
	{
		return false;
	}

	unsigned int numberOfPoints = 0;

	for (size_t i = 0; i < ply.GetPoints().size() / 3; i++)
	{
		auto x = ply.GetPoints()[i * 3];
		auto y = ply.GetPoints()[i * 3 + 1];
		auto z = ply.GetPoints()[i * 3 + 2];

		if (roi.contains(Eigen::Vector3f(x, y, z))) numberOfPoints++;
	}

	Initialize(numberOfPoints);

	unsigned int bufferIndex = 0;
	for (size_t i = 0; i < ply.GetPoints().size() / 3; i++)
	{
		auto x = ply.GetPoints()[i * 3];
		auto y = ply.GetPoints()[i * 3 + 1];
		auto z = ply.GetPoints()[i * 3 + 2];

		if(false == roi.contains(Eigen::Vector3f(x,y,z))) continue;

		auto nx = ply.GetNormals()[i * 3];
		auto ny = ply.GetNormals()[i * 3 + 1];
		auto nz = ply.GetNormals()[i * 3 + 2];

		auto r = ply.GetColors()[i * 3];
		auto g = ply.GetColors()[i * 3 + 1];
		auto b = ply.GetColors()[i * 3 + 2];

		h_buffers.positions[bufferIndex] = Eigen::Vector3f(x, y, z);
		h_buffers.normals[bufferIndex] = Eigen::Vector3f(nx, ny, nz);
		h_buffers.colors[bufferIndex] = Eigen::Vector3b(r * 255.0f, g * 255.0f, b * 255.0f);

		h_buffers.aabb.extend(Eigen::Vector3f(x, y, z));

		bufferIndex++;
	}

	HtoD();

	hashmap.InsertPoints(d_buffers);

	return true;
}

bool PointCloud::SaveToPLY(const std::string& filename)
{
	PLYFormat ply;

	for (size_t i = 0; i < h_buffers.numberOfPoints; i++)
	{
		ply.AddPointFloat3(h_buffers.positions[i].data());
		ply.AddNormalFloat3(h_buffers.normals[i].data());
		ply.AddColor(h_buffers.colors[i].x(), h_buffers.colors[i].y(), h_buffers.colors[i].z());
	}

	ply.Serialize(filename);

	return true;
}

bool PointCloud::LoadFromALP(const std::string& filename)
{
	ALPFormat<PointPNC> alp;
	if (false == alp.Deserialize(filename))
	{
		return false;
	}

	//printf("min: %f, %f, %f\n", get<0>(alp.GetAABBMin()), get<1>(alp.GetAABBMin()), get<2>(alp.GetAABBMin()));
	//printf("max: %f, %f, %f\n", get<0>(alp.GetAABBMax()), get<1>(alp.GetAABBMax()), get<2>(alp.GetAABBMax()));

	Initialize(alp.GetPoints().size());

	for (size_t i = 0; i < alp.GetPoints().size(); i++)
	{
		auto& p = alp.GetPoints()[i];

		h_buffers.positions[i] = Eigen::Vector3f(p.position.x, p.position.y, p.position.z);
		h_buffers.normals[i] = Eigen::Vector3f(p.normal.x, p.normal.y, p.normal.z);
		h_buffers.colors[i] = Eigen::Vector3b(p.color.x * 255.0f, p.color.y * 255.0f, p.color.z * 255.0f);

		h_buffers.aabb.extend(Eigen::Vector3f(p.position.x, p.position.y, p.position.z));
	}

	HtoD();

	hashmap.InsertPoints(d_buffers);

	//hashmap.SerializeToPLY("../../res/test.ply");

	return true;
}

bool PointCloud::LoadFromALP(const string& filename, const Eigen::AlignedBox3f& roi)
{
	ALPFormat<PointPNC> alp;
	if (false == alp.Deserialize(filename))
	{
		return false;
	}

	unsigned int numberOfPoints = 0;

	for (size_t i = 0; i < alp.GetPoints().size(); i++)
	{
		auto& p = alp.GetPoints()[i];

		if (roi.contains(Eigen::Vector3f(p.position.x, p.position.y, p.position.z))) numberOfPoints++;
	}

	Initialize(numberOfPoints);

	unsigned int bufferIndex = 0;
	for (size_t i = 0; i < alp.GetPoints().size(); i++)
	{
		auto& p = alp.GetPoints()[i];

		if (false == roi.contains(Eigen::Vector3f(p.position.x, p.position.y, p.position.z))) continue;

		h_buffers.positions[bufferIndex] = Eigen::Vector3f(p.position.x, p.position.y, p.position.z);
		h_buffers.normals[bufferIndex] = Eigen::Vector3f(p.normal.x, p.normal.y, p.normal.z);
		h_buffers.colors[bufferIndex] = Eigen::Vector3b(p.color.x * 255.0f, p.color.y * 255.0f, p.color.z * 255.0f);

		h_buffers.aabb.extend(Eigen::Vector3f(p.position.x, p.position.y, p.position.z));

		bufferIndex++;
	}

	HtoD();

	hashmap.InsertPoints(d_buffers);

	//hashmap.SerializeToPLY("../../res/test.ply");

	return true;
}

bool PointCloud::SaveToALP(const std::string& filename)
{
	ALPFormat<PointPNC> alp;
	
	for (size_t i = 0; i < h_buffers.numberOfPoints; i++)
	{
		PointPNC p;
		p.position = make_float3(h_buffers.positions[i].x(), h_buffers.positions[i].y(), h_buffers.positions[i].z());
		p.normal = make_float3(h_buffers.normals[i].x(), h_buffers.normals[i].y(), h_buffers.normals[i].z());
		p.color = make_float3((float)h_buffers.colors[i].x() / 255.0f, (float)h_buffers.colors[i].y() / 255.0f, (float)h_buffers.colors[i].z() / 255.0f);
		
		alp.AddPoint(p);
	}

	alp.Serialize(filename);

	return true;
}

__global__ void Kernel_ComputeVoxelNormalPCA(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

	auto centerPosition = Eigen::Vector3f(
		coord.x * info.voxelSize,
		coord.y * info.voxelSize,
		coord.z * info.voxelSize);

	Eigen::Matrix3f cov = Eigen::Matrix3f::Zero();
	unsigned int neighborCount = 0;
#pragma unroll
	for (int ni = 0; ni < 124; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_124[ni].x,
			coord.y + neighbor_offsets_124[ni].y,
			coord.z + neighbor_offsets_124[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		if (neighborVoxel == nullptr) continue;

		auto neighborPosition = Eigen::Vector3f(
			neighborCoord.x * info.voxelSize,
			neighborCoord.y * info.voxelSize,
			neighborCoord.z * info.voxelSize);

		Eigen::Vector3f d = neighborPosition - centerPosition;
		cov += d * d.transpose();
		neighborCount++;
	}

	cov /= (float)neighborCount;
	Eigen::SelfAdjointEigenSolver<Eigen::Matrix3f> solver(cov);
	centerVoxel->normal = solver.eigenvectors().col(0);
}

void PointCloud::ComputeVoxelNormalPCA()
{
	nvtxRangePushA("Compute Voxel Normal PCA");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeVoxelNormalPCA << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();

	nvtxRangePop();
}

__global__ void Kernel_ComputeVoxelNormalAverage(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

	Eigen::Vector3f normal = Eigen::Vector3f::Zero();
	unsigned int neighborCount = 0;
#pragma unroll
	for (int ni = 0; ni < 124; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_124[ni].x,
			coord.y + neighbor_offsets_124[ni].y,
			coord.z + neighbor_offsets_124[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		if (neighborVoxel == nullptr) continue;

		auto neighborNormal = (neighborVoxel->normal / (float)neighborVoxel->pointCount).normalized();

		normal += neighborNormal;
		neighborCount++;
	}

	centerVoxel->normal = normal / (float)neighborCount++;
}

void PointCloud::ComputeVoxelNormalAverage()
{
	nvtxRangePushA("Compute Voxel Normal Average");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeVoxelNormalAverage << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();

	nvtxRangePop();
}

__global__ void Kernel_SerializeVoxels(HashMapInfo info, PointCloudBuffers buffers)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr) return;

	auto position = Eigen::Vector3f(coord.x * info.voxelSize, coord.y * info.voxelSize, coord.z * info.voxelSize);
	auto normal = (voxel->normal / (float)voxel->pointCount).normalized();
	auto color = voxel->color;

	buffers.positions[threadid] = position;
	buffers.normals[threadid] = normal;
	buffers.colors[threadid] = color;
}

void PointCloud::SerializeVoxels(PointCloudBuffers& d_tempBuffers)
{
	nvtxRangePushA("SerializeVoxels");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_SerializeVoxels << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}

__global__ void Kernel_SerializeVoxelsColoringByLabel(HashMapInfo info, PointCloudBuffers buffers)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr) return;

	auto position = Eigen::Vector3f(coord.x * info.voxelSize, coord.y * info.voxelSize, coord.z * info.voxelSize);
	auto normal = (voxel->normal / (float)voxel->pointCount).normalized();
	
	float r = hashToFloat(voxel->label * 3 + 0);
	float g = hashToFloat(voxel->label * 3 + 1);
	float b = hashToFloat(voxel->label * 3 + 2);

	auto color = Eigen::Vector3b(r * 255.0f, g * 255.0f, b * 255.0f);

	buffers.positions[threadid] = position;
	buffers.normals[threadid] = normal;
	buffers.colors[threadid] = color;
}

void PointCloud::SerializeVoxelsColoringByLabel(PointCloudBuffers& d_tempBuffers)
{
	nvtxRangePushA("SerializeVoxelsColoringByLabel");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_SerializeVoxelsColoringByLabel << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}






__global__ void Kernel_ComputeNeighborCount(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

#pragma unroll
	for (int ni = 0; ni < 26; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_26[ni].x,
			coord.y + neighbor_offsets_26[ni].y,
			coord.z + neighbor_offsets_26[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		//if (neighborVoxel && neighborVoxel->label != 0)
		//{
		//	centerVoxel->neighborCount++;
		//}

		if (neighborVoxel == nullptr || neighborVoxel->label == 0)
		{
			centerVoxel->neighborCount++;
		}
	}
}

void PointCloud::ComputeNeighborCount()
{
	nvtxRangePushA("NeighborCount");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeNeighborCount << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();
}

__global__ void Kernel_SerializeColoringByNeighborCount(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t slot = GetVoxelSlot(info, coord);
	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr || voxel->label == 0) return;

	buffers.positions[idx] = voxel->position;
	buffers.normals[idx] = voxel->normal;

	//const Eigen::Vector3b COLORS[26] = {
	//{255, 0, 0},
	//{233, 0, 21},
	//{212, 0, 42},
	//{191, 0, 63},
	//{170, 0, 85},
	//{148, 0, 106},
	//{127, 0, 127},
	//{106, 0, 148},
	//{85, 0, 170},
	//{63, 0, 191},
	//{42, 0, 212},
	//{21, 0, 233},
	//{0, 0, 255},
	//{0, 21, 233},
	//{0, 42, 212},
	//{0, 63, 191},
	//{0, 85, 170},
	//{0, 106, 148},
	//{0, 127, 127},
	//{0, 148, 106},
	//{0, 170, 85},
	//{0, 191, 63},
	//{0, 212, 42},
	//{0, 233, 21},
	//{0, 255, 0},
	//{0, 255, 0}  // ������ �ߺ��� ���� ������ (���û���)
	//};

	//buffers.colors[idx] = COLORS[25 - voxel->neighborCount];

	if (19 <= voxel->neighborCount && voxel->neighborCount <= 25)
	{
		buffers.colors[idx] = Eigen::Vector3b(255, 0, 0);
	}
	else
	{
		buffers.colors[idx] = Eigen::Vector3b(100, 100, 100);
	}
}

void PointCloud::SerializeColoringByNeighborCount(PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByNeighborCount << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}





__global__ void Kernel_ClearNormalDiscontinuity(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	auto voxel = GetVoxel(info, slot);

	voxel->normalDiscontinue = 0;
}

__global__ void Kernel_ComputeNormalDiscontinuity(HashMapInfo info, float normalDiscontinuityThreshold)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

	auto centerNormal = (centerVoxel->normal / (float)centerVoxel->pointCount).normalized();

#pragma unroll
	for (int ni = 0; ni < 26; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_26[ni].x,
			coord.y + neighbor_offsets_26[ni].y,
			coord.z + neighbor_offsets_26[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		if (neighborVoxel == nullptr) continue;

		auto neighborNormal = (neighborVoxel->normal / (float)neighborVoxel->pointCount).normalized();

		if (cosf(normalDiscontinuityThreshold * M_PI / 180.0f) < centerNormal.dot(neighborNormal)) continue;

		centerVoxel->normalDiscontinue = 1;
		
		return;
	}
}

void PointCloud::ComputeNormalDiscontinuity(float normalDiscontinuityThreshold)
{
	nvtxRangePushA("NormalDiscontinuity");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_ClearNormalDiscontinuity << <gridOccupied, blockSize >> > (hashmap.info);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_ComputeNormalDiscontinuity << <gridOccupied, blockSize >> > (hashmap.info, normalDiscontinuityThreshold);

		hipDeviceSynchronize();
	}
}

__global__ void Kernel_SerializeColoringByNormalDiscontinuity(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t slot = GetVoxelSlot(info, coord);
	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr || voxel->label == 0) return;

	if (0 != voxel->normalDiscontinue)
	{
		buffers.colors[idx] = Eigen::Vector3b(255, 0, 0);
	}
	else
	{
		buffers.colors[idx] = Eigen::Vector3b(100, 100, 100);
	}
}

void PointCloud::SerializeColoringByNormalDiscontinuity(PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByNormalDiscontinuity << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}







__global__ void Kernel_ComputeNormalGradient(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

	int count = 0;
	centerVoxel->gradient = Eigen::Vector3f(0.0f, 0.0f, 0.0f);

#pragma unroll
	for (int ni = 0; ni < 26; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_26[ni].x,
			coord.y + neighbor_offsets_26[ni].y,
			coord.z + neighbor_offsets_26[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		if (neighborVoxel == nullptr || 0 == neighborVoxel->label) continue;

		centerVoxel->gradient +=
			(neighborVoxel->normal / (float)neighborVoxel->pointCount).normalized() -
			(centerVoxel->normal / (float)centerVoxel->pointCount).normalized();

		count++;
	}

	if (0 < count)
	{
		centerVoxel->gradient /= (float)count;
		//printf("%f, %f, %f\n", centerVoxel->gradient.x(), centerVoxel->gradient.y(), centerVoxel->gradient.z());
	}
}

void PointCloud::ComputeNormalGradient()
{
	nvtxRangePushA("ComputeNormalGradient");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;
	
	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeNormalGradient << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();
}

__global__ void Kernel_SerializeColoringByNormalGradient(HashMapInfo info, PointCloudBuffers buffers, float threshold)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t slot = GetVoxelSlot(info, coord);
	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr || voxel->label == 0) return;

	buffers.positions[idx] = voxel->position;
	buffers.normals[idx] = voxel->normal;
	auto g = voxel->gradient.normalized();

	float length = voxel->gradient.norm();
	if (length > threshold)
	{
		buffers.colors[idx] = Eigen::Vector3b(255, 0, 0);
	}
	else
	{
		buffers.colors[idx] = Eigen::Vector3b(100, 100, 100);
	}
}

void PointCloud::SerializeColoringByNormalGradient(float threshold, PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByNormalGradient << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers, threshold);

	hipDeviceSynchronize();
}







__global__ void Kernel_ComputeNormalDivergence(HashMapInfo info)
{
	unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	auto voxelSlot = GetVoxelSlot(info, coord);
	if (voxelSlot == UINT64_MAX) return;
	auto voxel = GetVoxel(info, voxelSlot);
	if (voxel == nullptr || voxel->label == 0) return;

	const Eigen::Vector3f n0 = voxel->normal.normalized();

	const int3 offsets[6] = {
		{1,0,0}, {-1,0,0}, {0,1,0}, {0,-1,0}, {0,0,1}, {0,0,-1}
	};

	float divergenceSum = 0.0f;
	int validNeighbors = 0;

	for (int ni = 0; ni < 6; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + offsets[ni].x,
			coord.y + offsets[ni].y,
			coord.z + offsets[ni].z);

		auto neighborVoxelSlot = GetVoxelSlot(info, neighborCoord);
		if (neighborVoxelSlot == UINT64_MAX) continue;

		auto neighborVoxel = GetVoxel(info, neighborVoxelSlot);
		if (neighborVoxel == nullptr || neighborVoxel->label == 0) continue;

		Eigen::Vector3f n1 = neighborVoxel->normal.normalized();
		float dot = fminf(fmaxf(n0.dot(n1), -1.0f), 1.0f);
		float angle = acosf(dot); // radians
		divergenceSum += angle;
		++validNeighbors;
	}

	voxel->divergence = (validNeighbors > 0) ? (divergenceSum / validNeighbors) : 0.0f;

	//printf("voxel->divergence : %f\n", voxel->divergence);
}

void PointCloud::ComputeNormalDivergence()
{
	nvtxRangePushA("ComputeNormalDivergence");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;
	
	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeNormalDivergence << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();
}

__global__ void Kernel_SerializeColoringByNormalDivergence(HashMapInfo info, PointCloudBuffers buffers, float threshold)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t slot = GetVoxelSlot(info, coord);
	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr || voxel->label == 0) return;

	buffers.positions[idx] = voxel->position;
	buffers.normals[idx] = voxel->normal;

	//printf("voxel->divergence : %f\n", voxel->divergence);

	if (voxel->divergence > threshold)
	{
		buffers.colors[idx] = Eigen::Vector3b(255, 0, 0);
	}
	else
	{
		buffers.colors[idx] = Eigen::Vector3b(100, 100, 100);
	}
}

void PointCloud::SerializeColoringByNormalDivergence(float threshold, PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByNormalDivergence << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers, threshold);

	hipDeviceSynchronize();
}







__global__ void Kernel_ComputeColorMultiplication(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	if (centerVoxel == nullptr || centerVoxel->label == 0) return;

#pragma unroll
	for (int ni = 0; ni < 26; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_26[ni].x,
			coord.y + neighbor_offsets_26[ni].y,
			coord.z + neighbor_offsets_26[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);

		if (neighborVoxel == nullptr || neighborVoxel->label == 0) return;

		float distance = std::sqrt(
			static_cast<float>(neighborVoxel->color.x()) * centerVoxel->color.x() +
			static_cast<float>(neighborVoxel->color.y()) * centerVoxel->color.y() +
			static_cast<float>(neighborVoxel->color.z()) * centerVoxel->color.z());

		//printf("distance : %f\n", distance);

		if (centerVoxel->colorDistance < distance) centerVoxel->colorDistance = distance;
	}
}

void PointCloud::ComputeColorMultiplication()
{
	nvtxRangePushA("ComputeColorMultiplication");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;
	
	unsigned int blockSize = 256;
	unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

	Kernel_ComputeColorMultiplication << <gridOccupied, blockSize >> > (hashmap.info);

	hipDeviceSynchronize();
}

__global__ void Kernel_SerializeColoringByColorMultiplication(HashMapInfo info, PointCloudBuffers buffers, float threshold)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t slot = GetVoxelSlot(info, coord);
	HashMapVoxel* voxel = GetVoxel(info, slot);
	if (voxel == nullptr || voxel->label == 0) return;

	buffers.positions[idx] = voxel->position;
	buffers.normals[idx] = voxel->normal;

	if (voxel->colorDistance > threshold)
	{
		buffers.colors[idx] = Eigen::Vector3b(255, 0, 0);
	}
	else
	{
		buffers.colors[idx] = Eigen::Vector3b(100, 100, 100);
	}
}

void PointCloud::SerializeColoringByColorMultiplication(float threshold, PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByColorMultiplication << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers, threshold);

	hipDeviceSynchronize();
}







__device__ __forceinline__ unsigned int FindRootVoxel(HashMapInfo info, unsigned int idx)
{
	while (info.d_hashTable[idx].label != idx)
	{
		unsigned int parent = info.d_hashTable[idx].label;
		unsigned int grandparent = info.d_hashTable[parent].label;

		auto& voxelA = info.d_hashTable[idx];
		auto& voxelB = info.d_hashTable[parent];

		//// �Ÿ� ����
		//Eigen::Vector3f centerA = voxelA.position / voxelA.pointCount;
		//Eigen::Vector3f centerB = voxelB.position / voxelB.pointCount;
		//float distSq = (centerA - centerB).squaredNorm();
		//if (distSq > kMaxMergeDistance * kMaxMergeDistance) break;

		//// ���� ����
		//Eigen::Vector3f nA = (voxelA.normal / voxelA.pointCount).normalized();
		//Eigen::Vector3f nB = (voxelB.normal / voxelB.pointCount).normalized();
		//if (nA.dot(nB) < kMinNormalDot) break;

		// ��� ����
		if (parent != grandparent)
			info.d_hashTable[idx].label = grandparent;

		idx = info.d_hashTable[idx].label;
	}
	return idx;
}

__device__ __forceinline__ void UnionVoxel(HashMapInfo info, unsigned int a, unsigned int b)
{
	unsigned int rootA = FindRootVoxel(info, a);
	unsigned int rootB = FindRootVoxel(info, b);
	if (rootA == rootB) return;

	auto& voxelA = info.d_hashTable[rootA];
	auto& voxelB = info.d_hashTable[rootB];

	//auto dx = abs(voxelA.coord.x - voxelB.coord.x);
	//auto dy = abs(voxelA.coord.y - voxelB.coord.y);
	//auto dz = abs(voxelA.coord.z - voxelB.coord.z);
	//if (dx >= 2 || dy >= 2 || dz >= 2)
	//{
	//	printf("??????????\n");
	//	return;
	//}



	//// �߽� ��ǥ ���
	//Eigen::Vector3f centerA = voxelA.position / voxelA.pointCount;
	//Eigen::Vector3f centerB = voxelB.position / voxelB.pointCount;
	//float distSq = (centerA - centerB).squaredNorm();

	//if (distSq > kMaxMergeDistance * kMaxMergeDistance) return;

	//// ���� ���絵 �Ǵ�
	//Eigen::Vector3f nA = (voxelA.normal / voxelA.pointCount).normalized();
	//Eigen::Vector3f nB = (voxelB.normal / voxelB.pointCount).normalized();
	//if (nA.dot(nB) < kMinNormalDot) return;

	// ����
	if (rootA < rootB)
		atomicMin(&voxelB.label, rootA);
	else
		atomicMin(&voxelA.label, rootB);
}

__global__ void Kernel_ClearLabels(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;
	
	auto voxel = GetVoxel(info, slot);

	voxel->label = slot;
}

__global__ void Kernel_InterVoxelHashMerge26Way(HashMapInfo info, float normalDegreeThreshold)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];
	size_t slot = GetVoxelSlot(info, coord);
	if (slot == UINT64_MAX) return;

	HashMapVoxel* centerVoxel = GetVoxel(info, slot);
	//if (centerVoxel == nullptr || centerVoxel->label == 0) return;
	if (centerVoxel == nullptr) return;

	auto centerNormal = (centerVoxel->normal / (float)centerVoxel->pointCount).normalized();

#pragma unroll
	for (int ni = 0; ni < 26; ++ni)
	{
		int3 neighborCoord = make_int3(
			coord.x + neighbor_offsets_26[ni].x,
			coord.y + neighbor_offsets_26[ni].y,
			coord.z + neighbor_offsets_26[ni].z);

		size_t neighborSlot = GetVoxelSlot(info, neighborCoord);
		HashMapVoxel* neighborVoxel = GetVoxel(info, neighborSlot);
 
		//if (neighborVoxel && neighborVoxel->label != 0)
		if (neighborVoxel)
		{
			auto neighborNormal = (neighborVoxel->normal / (float)neighborVoxel->pointCount).normalized();

			if (cosf(normalDegreeThreshold * M_PI / 180.0f) > centerNormal.dot(neighborNormal)) continue;

			UnionVoxel(info, slot, neighborSlot);
		}
	}
}

__global__ void Kernel_CompressVoxelHashLabels(HashMapInfo info)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= *info.d_numberOfOccupiedVoxels) return;

	int3 coord = info.d_occupiedVoxelIndices[threadid];

	size_t h = voxel_hash(coord, info.capacity);
	for (int i = 0; i < info.maxProbe; ++i)
	{
		size_t probe = (h + i) % info.capacity;
		auto& voxel = info.d_hashTable[probe];

		if (0 == voxel.label) break;

		voxel.label = FindRootVoxel(info, voxel.label);
	}
}

__global__ void Kernel_GetLabels(HashMapInfo info, Eigen::Vector3f* points, unsigned int* labels, unsigned int numberOfPoints)
{
	unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadid >= numberOfPoints) return;

	auto& p = points[threadid];
	int3 coord = make_int3(
		floorf(p.x() / info.voxelSize),
		floorf(p.y() / info.voxelSize),
		floorf(p.z() / info.voxelSize));

	size_t h = voxel_hash(coord, info.capacity);
	labels[threadid] = UINT_MAX;

	for (int i = 0; i < info.maxProbe; ++i)
	{
		size_t probe = (h + i) % info.capacity;
		if (info.d_hashTable[probe].label == 0) break;

		auto& voxel = info.d_hashTable[probe];

		if (voxel.coord.x == coord.x &&
			voxel.coord.y == coord.y &&
			voxel.coord.z == coord.z)
		{
			labels[threadid] = voxel.label;
			return;
		}
	}
}

vector<unsigned int> PointCloud::Clustering(float normalDegreeThreshold)
{
	nvtxRangePushA("Clustering");

	unsigned int numberOfOccupiedVoxels = hashmap.info.h_numberOfOccupiedVoxels;
	
	vector<unsigned int> labels;

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_ClearLabels << <gridOccupied, blockSize >> > (hashmap.info);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_InterVoxelHashMerge26Way << <gridOccupied, blockSize >> > (hashmap.info, normalDegreeThreshold);

		hipDeviceSynchronize();
	}

	{
		unsigned int blockSize = 256;
		unsigned int gridOccupied = (numberOfOccupiedVoxels + blockSize - 1) / blockSize;

		Kernel_CompressVoxelHashLabels << <gridOccupied, blockSize >> > (hashmap.info);

		hipDeviceSynchronize();
	}

	labels.resize(h_buffers.numberOfPoints);
	{
		unsigned int* d_labels = nullptr;
		hipMalloc(&d_labels, sizeof(unsigned int) * h_buffers.numberOfPoints);

		unsigned int blockSize = 256;
		unsigned int gridOccupied = (h_buffers.numberOfPoints + blockSize - 1) / blockSize;

		Kernel_GetLabels << <gridOccupied, blockSize >> > (hashmap.info, d_buffers.positions, d_labels, h_buffers.numberOfPoints);

		hipDeviceSynchronize();

		hipMemcpy(labels.data(), d_labels, sizeof(unsigned int) * h_buffers.numberOfPoints, hipMemcpyDeviceToHost);

		hipFree(d_labels);
	}

	nvtxRangePop();

	return labels;
}

__global__ void Kernel_SerializeColoringByLabel(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t h = voxel_hash(coord, info.capacity);

	for (unsigned int i = 0; i < info.maxProbe; ++i)
	{
		size_t slot = (h + i) % info.capacity;
		auto& voxel = info.d_hashTable[slot];

		if (0 == voxel.label) return;

		if (voxel.coord.x == coord.x &&
			voxel.coord.y == coord.y &&
			voxel.coord.z == coord.z)
		{
			buffers.positions[idx] = voxel.position;
			buffers.normals[idx] = voxel.normal;

			float r = hashToFloat(voxel.label * 3 + 0);
			float g = hashToFloat(voxel.label * 3 + 1);
			float b = hashToFloat(voxel.label * 3 + 2);

			buffers.colors[idx] = Eigen::Vector3b(r * 255.0f, g * 255.0f, b * 255.0f);

			return;
		}
	}
}

void PointCloud::SerializeColoringByLabel(PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SerializeColoringByLabel << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}






__global__ void Kernel_SplitByNormal(HashMapInfo info, PointCloudBuffers buffers)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= buffers.numberOfPoints) return;

	auto& p = buffers.positions[idx];
	int3 coord = make_int3(floorf(p.x() / info.voxelSize), floorf(p.y() / info.voxelSize), floorf(p.z() / info.voxelSize));
	size_t h = voxel_hash(coord, info.capacity);

	for (unsigned int i = 0; i < info.maxProbe; ++i)
	{
		size_t slot = (h + i) % info.capacity;
		auto& voxel = info.d_hashTable[slot];

		if (0 == voxel.label) return;

		if (voxel.coord.x == coord.x &&
			voxel.coord.y == coord.y &&
			voxel.coord.z == coord.z)
		{
			buffers.positions[idx] = voxel.position;
			buffers.normals[idx] = voxel.normal;

			float r = hashToFloat(voxel.label * 3 + 0);
			float g = hashToFloat(voxel.label * 3 + 1);
			float b = hashToFloat(voxel.label * 3 + 2);

			buffers.colors[idx] = Eigen::Vector3b(r * 255.0f, g * 255.0f, b * 255.0f);

			return;
		}
	}
}

void PointCloud::SplitByNormal(PointCloudBuffers& d_tempBuffers)
{
	d_buffers.CopyTo(d_tempBuffers);

	unsigned int blockSize = 256;
	unsigned int gridOccupied = (d_buffers.numberOfPoints + blockSize - 1) / blockSize;

	Kernel_SplitByNormal << <gridOccupied, blockSize >> > (hashmap.info, d_tempBuffers);

	hipDeviceSynchronize();
}
